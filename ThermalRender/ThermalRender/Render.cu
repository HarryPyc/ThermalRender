#include "hip/hip_runtime.h"
#include "Render.cuh"
#include "Mesh.h"
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include "Camera.cuh"
#include <glm/gtc/constants.hpp>
#include "ThermalData.h"

#define STB_IMAGE_IMPLEMENTATION
#include <stb_image.h>


using namespace glm;

texture<float4, hipTextureType2D, hipReadModeElementType> emisMap, normalMap;

__constant__ MeshInfo m;
__constant__ Object objList[10];
__constant__ Wave wave_sky, wave_zero;
__constant__ int w, h, MAX_DEPTH = 6, d_Samples;
__constant__ hiprandState_t* state;
__constant__ Camera cam;
__constant__ float EPSILON = 1e-4, float PI;

__global__ void cuRand_Setup_Kernel(int seed) {
	const int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x >= w || y >= h) return;

	hiprand_init(seed, x + y * w, 0, &state[x + y * w]);
}

void initTexture(textureReference& tex, const char* path) {
	int w, h, comp;
	float* h_image = stbi_loadf(path, &w, &h, &comp, 4);

	hipChannelFormatDesc format = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	
	hipArray_t cuArray;
	gpuErrchk(hipMallocArray(&cuArray, &format, w, h));
	gpuErrchk(hipMemcpyToArray(cuArray, 0, 0, h_image, w * h * 4 * sizeof(float), hipMemcpyHostToDevice));

	tex.addressMode[0] = hipAddressModeWrap;
	tex.addressMode[1] = hipAddressModeWrap;
	tex.filterMode = hipFilterModeLinear;
	tex.normalized = true;

	gpuErrchk(hipBindTextureToArray(&tex, cuArray, &format));
	delete[] h_image;
}

void initRender(int width, int height) {
	std::vector<Object> h_obj;
	MeshInfo h_m = initMesh(h_obj);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(objList), h_obj.data(), h_obj.size() * sizeof(Object)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(m), &h_m, sizeof(MeshInfo)));

	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(w), &width, sizeof(unsigned)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(h), &height, sizeof(unsigned)));

	hiprandState_t* d_randState;
	gpuErrchk(hipMalloc((void**)&d_randState, width * height * sizeof(hiprandState_t)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(state), &d_randState, sizeof(d_randState)));

	srand(time(0));
	int seed = rand();
	dim3 blockDim(16, 16, 1), gridDim(width / blockDim.x + 1, height / blockDim.y + 1, 1);
	cuRand_Setup_Kernel << < gridDim, blockDim >> > (seed);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	//Setup Camera
	Camera h_cam;
	h_cam.init(vec3(-1.f, 2.f, 1.f), vec3(0.f, 0.f, -5.f), vec3(0.f, 1.0f, 0.f));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cam), &h_cam, sizeof(Camera)));

	float h_pi = pi<float>();
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(PI), &h_pi, sizeof(float)));
	gpuErrchk(hipDeviceSetLimit(hipLimitStackSize, 1024 * 8));


	initTexture(normalMap, "asset/texture/cube_normal.jpg");

	Wave h_zero = Wave::GetWave(0.f), h_sky = GetSky();
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(wave_zero), &h_zero, sizeof(Wave)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(wave_sky), &h_sky, sizeof(Wave)));
}
struct Ray {
	vec3 o, d, invD;

	__device__ Ray(vec3 origin, vec3 dir) : o(origin), d(dir) {
		invD = 1.f / d;
	};
	__device__ bool RayTriangleIntersection(const vec3& v0, const vec3& v1, const vec3& v2, float &t, float& u, float& v) {
		vec3 edge1, edge2, h, s, q;
		float a, f;
		edge1 = v1 - v0;
		edge2 = v2 - v0;
		h = cross(d, edge2);
		a = dot(edge1, h);
		if (a > -EPSILON && a < EPSILON)
			return false;    // This ray is parallel to this triangle.
		f = 1.0 / a;
		s = o - v0;
		u = f * dot(s, h);
		if (u < 0.0 || u > 1.0)
			return false;
		q = cross(s, edge1);
		v = f * dot(d, q);
		if (v < 0.0 || u + v > 1.0)
			return false;
		// At this stage we can compute t to find out where the intersection point is on the line.
		t = f * dot(edge2, q);
		return t > EPSILON;
	}
	__device__ bool RayAABBIntersection(const glm::vec3& minAABB, const glm::vec3& maxAABB) {
		glm::vec3 t0s = (minAABB - o) * invD;
		glm::vec3 t1s = (maxAABB - o) * invD;

		glm::vec3 tsmaller = glm::min(t0s, t1s);
		glm::vec3 tbigger = glm::max(t0s, t1s);

		float tmin = glm::max(-9999.f, glm::max(tsmaller[0], glm::max(tsmaller[1], tsmaller[2])));
		float tmax = glm::min(9999.f, glm::min(tbigger[0], glm::min(tbigger[1], tbigger[2])));
		//t = (tmin + tmax) / 2.f;
		return (tmin < tmax) && tmax > 0.f;
	}
};

__device__ void FetchMesh(vec3& n, vec2& uv, int A, int B, int C, float u, float v) {
	
	vec3 n0(m.d_n[3 * A], m.d_n[3 * A + 1], m.d_n[3 * A + 2]), n1(m.d_n[3 * B], m.d_n[3 * B + 1], m.d_n[3 * B + 2]),
		n2(m.d_n[3 * C], m.d_n[3 * C + 1], m.d_n[3 * C + 2]);
	vec2 uv0(m.d_uv[2 * A], m.d_uv[2 * A + 1]), uv1(m.d_uv[2 * B], m.d_uv[2 * B + 1]), uv2(m.d_uv[2 * C], m.d_uv[2 * C + 1]);
	n = (1.f - u - v) * n0 + u * n1 + v * n2;
	n = normalize(n);

	uv = (1.f - u - v) * uv0 + u * uv1 + v * uv2;
	uv.y = 1.0f - uv.y;
}


__device__ Wave trace(Ray ray, int depth, hiprandState_t& state) {
	if (depth > MAX_DEPTH) return wave_zero;
	float u, v, t = 9999.f;
	int A, B, C, i_obj = -1;
	//Find the nearest triangle
	for (int i = 0; i < m.N; i++) {
		Object obj = objList[i];
		if (ray.RayAABBIntersection(obj.minAABB, obj.maxAABB)) {
			for (int j = 0; j < obj.N / 3; j++) {
				unsigned int idx0 = obj.d_idx[3 * j], idx1 = obj.d_idx[3 * j + 1], idx2 = obj.d_idx[3 * j + 2];
				vec3 v0(m.d_v[3 * idx0], m.d_v[3 * idx0 + 1], m.d_v[3 * idx0 + 2]), v1(m.d_v[3 * idx1], m.d_v[3 * idx1 + 1], m.d_v[3 * idx1 + 2]),
					v2(m.d_v[3 * idx2], m.d_v[3 * idx2 + 1], m.d_v[3 * idx2 + 2]);
				float _t, _u, _v;
				if (ray.RayTriangleIntersection(v0, v1, v2, _t, _u, _v) &&  _t < t) {
					t = _t, u = _u, v = _v;
					A = idx0, B = idx1, C = idx2;
					i_obj = i;
				}
			}
		}
	}

	if (i_obj == -1) return wave_sky;
	//Fetch vertex position, normal and texture coordinates
	const Object obj = objList[i_obj];
	vec3 p, n; vec2 uv;
	FetchMesh(n, uv, A, B, C, u, v);

	vec3 color = obj.color;
	if (obj.useTex) {
		float4 n_sample = tex2D(normalMap, uv.x, uv.y);

		memcpy(&n[0], &n_sample, 3 * sizeof(float));
		n = normalize(n * 2.0f - 1.0f);
	}
	p = ray.o + ray.d * t + EPSILON * n;

	if (obj.refl_type == 0)//Specular
	{
		vec3 r = reflect(ray.d, n);
		return obj.emis + obj.refl * trace(Ray(p, r), depth + 1, state);
	}
	else if (obj.refl_type == 1) //Diffuse;
	{
		vec3 a = normalize(abs(n.x) < 1 - EPSILON ? cross(vec3(1, 0, 0), n) : cross(vec3(0, 1, 0), n)), b = cross(a, n);
		float alpha = 2.f * PI * hiprand_uniform(&state), beta = hiprand_uniform(&state);
		vec3 newDir = (glm::cos(alpha ) * a + glm::sin(alpha) * b) * sqrt(1.f - beta * beta) + beta * n;

		return obj.emis + obj.refl * trace(Ray(p, newDir), depth + 1, state);
	}

}


__global__ void RayTracingKernel(float* d_pbo) {
	const int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x >= w || y >= h) return;
	const int idx = x + y * w;
	hiprandState_t localState = state[idx];//use register for local efficiency
	
	float u = float(x) + hiprand_uniform(&localState), v = float(y) + hiprand_uniform(&localState);//Anti-alising
	Ray ray(cam.pos, cam.UnProject(u / float(w), v / float(h)));
	Wave curWave = trace(ray, 0, localState);

	Wave preWave;
	memcpy(&preWave[0], d_pbo + 11 * idx, 11 * sizeof(float));
	curWave = (preWave * float(d_Samples - 1) + curWave) / float(d_Samples);
	memcpy(d_pbo + 11 * idx, &curWave[0], 11 * sizeof(float));
	state[idx] = localState;
}

void render(float* d_pbo, int _w, int _h, int& h_Samples)
{
	h_Samples++; 
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_Samples), &h_Samples, sizeof(int)));
	dim3 blockDim(16, 16, 1), gridDim(_w / blockDim.x + 1, _h / blockDim.y + 1, 1);
	RayTracingKernel << <gridDim, blockDim >> > (d_pbo);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());
}
